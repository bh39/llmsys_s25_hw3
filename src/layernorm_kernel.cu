#include "hip/hip_runtime.h"
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>
#include <cstddef>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32


/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size * seq_len, hidden_size], ln result.
vars: [batch_size * seq_len], variance per token
means: [batch_size * seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute x and x^2 with reinterpret_cast by casting to float4 for speedup
  // 2. Compute reduce sum with blockReduce and add epsilon with LN_EPSILON
  // 3. Compute layernorm result with reinterpret_cast by casting to float4 for speedup
  
  // Step 1: Compute sums of x and x^2
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;  
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum += val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // Step 2: reduction and compute statistics
  blockReduce<ReduceType::kSum, 1>(&l_sum);
  blockReduce<ReduceType::kSum, 1>(&l_square_sum);
  
  __shared__ float s_mean, s_variance;
  if (threadIdx.x == 0) {
    s_mean = l_sum / (hidden_size * 4);
    s_variance = l_square_sum / (hidden_size * 4) - s_mean * s_mean + LN_EPSILON;
    
    // Store mean and variance
    vars[blockIdx.x] = s_variance;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
  }
  __syncthreads();
  
  // inverse standard deviation for normalization
  float inv_std = rsqrtf(s_variance);
  
  // Step 3: get normalized result
  float4 *output_f4 = reinterpret_cast<float4 *>(ln_res) + blockIdx.x * hidden_size;
  const float4 *scale_f4 = reinterpret_cast<const float4 *>(scale);
  const float4 *bias_f4 = reinterpret_cast<const float4 *>(bias);
  
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    float4 scale_val = scale_f4[idx];
    float4 bias_val = bias_f4[idx];
    
    float4 result;
    // normalization: 
    result.x = scale_val.x * ((val.x - s_mean) * inv_std) + bias_val.x;
    result.y = scale_val.y * ((val.y - s_mean) * inv_std) + bias_val.y;
    result.z = scale_val.z * ((val.z - s_mean) * inv_std) + bias_val.z;
    result.w = scale_val.w * ((val.w - s_mean) * inv_std) + bias_val.w;
    
    output_f4[idx] = result;
  }
  /// END ASSIGN3_2
}

extern "C" {
void launch_layernorm(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  int float_size = sizeof(float);
  int input_size = batch_size * hidden_dim * float_size;
  int scale_size = hidden_dim * float_size;
  int bias_size = hidden_dim * float_size;
  int output_size = batch_size * hidden_dim * float_size;
  int mean_size = batch_size * float_size;
  int var_size = batch_size * float_size;


  float *d_ln_res, *d_vars, *d_means, *d_inp, *d_scale, *d_bias;
  hipMalloc((void **)&d_ln_res, output_size);
  hipMalloc((void **)&d_vars, var_size);
  hipMalloc((void **)&d_means, mean_size);
  hipMalloc((void **)&d_inp, input_size);
  hipMalloc((void **)&d_scale, scale_size);
  hipMalloc((void **)&d_bias, bias_size);

  hipMemcpy(d_inp, inp, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_scale, scale, scale_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, bias_size, hipMemcpyHostToDevice);

  // For using float4
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      d_ln_res, d_vars, d_means, d_inp, d_scale, d_bias, hidden_dim);

  // Copy back to the host
  hipMemcpy(ln_res, d_ln_res, output_size, hipMemcpyDeviceToHost);
  hipMemcpy(vars, d_vars, var_size, hipMemcpyDeviceToHost);
  hipMemcpy(means, d_means, mean_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm Error: %s\n", hipGetErrorString(err));
    // Handle the error (e.g., by exiting the program)
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_ln_res);
  hipFree(d_vars);
  hipFree(d_means);
  hipFree(d_inp);
  hipFree(d_scale);
  hipFree(d_bias);

}
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma

@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad,
                                        const T *inp, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, int rows, int width) {

  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute the partial gradients by looping across inp rows
  // 2. Store the partial gradients in the shared memory arrays
  // 3. Compute the reduce sum of the shared memory arrays with g.shfl_down
  //      -> More hints about `g.shfl_down`:
  //      -> https://developer.nvidia.com/blog/cooperative-groups/#:~:text=Using%20thread_block_tile%3A%3Ashfl_down()%20to%20simplify%20our%20warp%2Dlevel%20reduction%20does%20benefit%20our%20code%3A%20it%20simplifies%20it%20and%20eliminates%20the%20need%20for%20shared%20memory
  //      -> The highlighted line gives you a conceptual understanding of what the g.shfl_down is doing. Usually, the threads inside a block need to load everything to shared memory and work together to reduce the result (like what you have implemented in the hw1 for reduce function). 
  //      -> Now g.shfl_down helps you do so without consuming any shared memory. g.shfl_down makes it more efficient.
  // 4. Assign the final result to the correct position in the global output

  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  // Step 1:
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= width) return;
  
  float dbetta_sum = 0.f;
  float dgamma_sum = 0.f;
  
  for (int r = threadIdx.y; r < rows; r += blockDim.y) {
    int r_idx = r * width + idx;
    float out_grad_val = out_grad[r_idx];
    
    // xhat
    float xhat;
    if (means != nullptr) {
      // xhat = (inp - mean) * rsqrt(var)
      xhat = (inp[r_idx] - means[r]) * rsqrtf(vars[r]);
    } else {
      // xhat = (output - betta) / gamma
      xhat = (inp[r_idx] - betta[idx]) / gamma[idx];
    }
    
    dbetta_sum += out_grad_val;
    dgamma_sum += out_grad_val * xhat;
  }
  
  // Step 2: 
  betta_buffer[threadIdx.y][threadIdx.x] = dbetta_sum;
  gamma_buffer[threadIdx.y][threadIdx.x] = dgamma_sum;
  __syncthreads();
  
  // Step 3: 
  if (threadIdx.y == 0) {
    float dbetta_val = betta_buffer[threadIdx.x][threadIdx.x];
    float dgamma_val = gamma_buffer[threadIdx.x][threadIdx.x];
    
    for (int i = 1; i < TILE_DIM; i++) {
      dbetta_val += betta_buffer[i][threadIdx.x];
      dgamma_val += gamma_buffer[i][threadIdx.x];
    }
    
    // reduction within the warp using shfl_down
    for (int i = 1; i < TILE_DIM; i *= 2) {
      dbetta_val += g.shfl_down(dbetta_val, i);
      dgamma_val += g.shfl_down(dgamma_val, i);
    }
    
    // Step 4: 
    if (threadIdx.x == 0) {
      betta_grad[blockIdx.x] = dbetta_val;
      gamma_grad[blockIdx.x] = dgamma_val;
    }
  }
  /// END ASSIGN3_2
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad, const T *inp,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute dxhat=dy*w with reinterpret_cast by casting to float4 for speedup
  // 2. Compute xhat with reinterpret_cast by casting to float4 for speedup
  // 3. Compute reduce sum for dxhat and dxhat*xhat with blockReduce
  // 4. Compute final gradient
  
  // Step 1
  int idx = blockIdx.x;
  float4 *inp_grad_f4 = reinterpret_cast<float4 *>(inp_grad) + idx * hidden_dim;
  const float4 *out_grad_f4 = reinterpret_cast<const float4 *>(out_grad) + idx * hidden_dim;
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + idx * hidden_dim;
  const float4 *gamma_f4 = reinterpret_cast<const float4 *>(gamma);
  const float4 *betta_f4 = (betta != nullptr) ? reinterpret_cast<const float4 *>(betta) : nullptr;
  
  float sum_dxhat = 0.f;
  float sum_dxhat_xhat = 0.f;
  
  // For each thread, process a chunk of the hidden dimensions
  for (int tid = threadIdx.x; tid < hidden_dim; tid += blockDim.x) {
    float4 dout = out_grad_f4[tid];
    float4 g = gamma_f4[tid];
    
    // Compute dxhat = dout * gamma
    float4 dxhat;
    dxhat.x = dout.x * g.x;
    dxhat.y = dout.y * g.y;
    dxhat.z = dout.z * g.z;
    dxhat.w = dout.w * g.w;
    
    // Step 2: 
    float4 xhat;
    float4 i = inp_f4[tid];
    float mean = means ? means[idx] : 0.f;
    float inv_std = rsqrtf(vars[idx]);
    
    if (means != nullptr) {
      xhat.x = (i.x - mean) * inv_std;
      xhat.y = (i.y - mean) * inv_std;
      xhat.z = (i.z - mean) * inv_std;
      xhat.w = (i.w - mean) * inv_std;
    } else {
      float4 b = betta_f4[tid];
      xhat.x = (i.x - b.x) / g.x;
      xhat.y = (i.y - b.y) / g.y;
      xhat.z = (i.z - b.z) / g.z;
      xhat.w = (i.w - b.w) / g.w;
    }
    
    // Accumulate for reduction
    sum_dxhat += dxhat.x + dxhat.y + dxhat.z + dxhat.w;
    sum_dxhat_xhat += dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z + dxhat.w * xhat.w;
    
    inp_grad_f4[tid] = dxhat;
  }
  
  // Step 3: Compute reduce sum for dxhat and dxhat*xhat
  blockReduce<ReduceType::kSum, 1>(&sum_dxhat);
  blockReduce<ReduceType::kSum, 1>(&sum_dxhat_xhat);
  
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat, s_inv_std;
  
  if (threadIdx.x == 0) {
    s_sum_dxhat = sum_dxhat;
    s_sum_dxhat_xhat = sum_dxhat_xhat;
    s_inv_std = rsqrtf(vars[idx]);
  }
  __syncthreads();
  
  // Step 4:
  float m_inv = 1.0f / (hidden_dim * 4);  // inverse of hidden dimension
  
  for (int tid = threadIdx.x; tid < hidden_dim; tid += blockDim.x) {
    float4 dxhat = inp_grad_f4[tid];
    float4 i = inp_f4[tid];
    float mean = means ? means[idx] : 0.f;
    
    float4 xhat;
    if (means != nullptr) {
      xhat.x = (i.x - mean) * s_inv_std;
      xhat.y = (i.y - mean) * s_inv_std;
      xhat.z = (i.z - mean) * s_inv_std;
      xhat.w = (i.w - mean) * s_inv_std;
    } else {
      float4 g = gamma_f4[tid];
      float4 b = betta_f4[tid];
      xhat.x = (i.x - b.x) / g.x;
      xhat.y = (i.y - b.y) / g.y;
      xhat.z = (i.z - b.z) / g.z;
      xhat.w = (i.w - b.w) / g.w;
    }
    
    // Compute final gradient:
    float4 result;
    result.x = (dxhat.x - m_inv * (s_sum_dxhat + xhat.x * s_sum_dxhat_xhat)) * s_inv_std;
    result.y = (dxhat.y - m_inv * (s_sum_dxhat + xhat.y * s_sum_dxhat_xhat)) * s_inv_std;
    result.z = (dxhat.z - m_inv * (s_sum_dxhat + xhat.z * s_sum_dxhat_xhat)) * s_inv_std;
    result.w = (dxhat.w - m_inv * (s_sum_dxhat + xhat.w * s_sum_dxhat_xhat)) * s_inv_std;
    
    inp_grad_f4[tid] = result;
  }
  
  assert(false && "Not Implemented");
  /// END ASSIGN3_2
}
extern "C" {
void launch_layernorm_bw(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *inp, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch_size, int hidden_dim,
                         hipStream_t stream_1, hipStream_t stream_2) {
  
  // Allocate device memory
  float *d_gamma_grad, *d_betta_grad, *d_inp_grad, *d_out_grad, *d_inp, *d_gamma, *d_betta, *d_vars, *d_means;
  int grad_output_size = batch_size * hidden_dim * sizeof(float);
  int gamma_betta_size = hidden_dim * sizeof(float);
  int vars_means_size = batch_size * sizeof(float);

  hipMalloc((void **)&d_gamma_grad, gamma_betta_size);
  hipMalloc((void **)&d_betta_grad, gamma_betta_size);
  hipMalloc((void **)&d_inp_grad, grad_output_size);
  hipMalloc((void **)&d_out_grad, grad_output_size);
  hipMalloc((void **)&d_inp, grad_output_size);
  hipMalloc((void **)&d_gamma, gamma_betta_size);
  hipMalloc((void **)&d_betta, gamma_betta_size);
  hipMalloc((void **)&d_vars, vars_means_size);
  hipMalloc((void **)&d_means, vars_means_size);

  // Copy memory to device
  hipMemcpy((void *)d_out_grad, out_grad, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_inp, inp, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_gamma, gamma, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_betta, betta, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_vars, vars, vars_means_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_means, means, vars_means_size, hipMemcpyHostToDevice);

  // Launch kernels
  // Compute grad of gamma and betta
  // This calculates the number of blocks needed to cover the data along the specified dimension, rounds it up.
  // The result is then multiplied by TILE_DIM to ensure that the grid size is a multiple of TILE_DIM.
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream_1>>>(
      d_gamma_grad, d_betta_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars,
      d_means, batch_size, hidden_dim);

  // Compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch_size, nthread, 0, stream_2>>>(
      d_inp_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars, d_means, hidden_dim);

  // Synchronize and check for errors
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy back to host
  hipMemcpy(gamma_grad, d_gamma_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(betta_grad, d_betta_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(inp_grad, d_inp_grad, grad_output_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_gamma_grad);
  hipFree(d_betta_grad);
  hipFree(d_inp_grad);
  hipFree((void *)d_out_grad);
  hipFree((void *)d_inp);
  hipFree((void *)d_gamma);
  hipFree((void *)d_betta);
  hipFree((void *)d_vars);
  hipFree((void *)d_means);
}}
}}
